/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void strongestNeighborScan_gpu(int * src, int * oldDst, int * newDst, int * oldWeight, int * newWeight, int * madeChanges, int distance, int numEdges) {
	/*YOUR CODE HERE*/
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//this is the thread id and will be used as index
	while( tid < numEdges )//as long as thread id is in bounds of the array
	{
		if( tid - distance >= 0 )//then we can do something, it's in bounds of the array
		{
			if( src[tid - distance] == src[tid] )//check if the two are in the same segment
			{
				if( oldWeight[tid - distance] >= oldWeight[tid] )//check the max, and update accordingly
				{
					newDst[tid] = oldDst[tid - distance];
					newWeight[tid] = oldWeight[tid - distance];
					(*madeChanges) = 1;
				}
				else
				{
					newDst[tid] = oldDst[tid];
					newWeight[tid] = oldWeight[tid];
				}
			}
			else
			{
				newDst[tid] = oldDst[tid];
				newWeight[tid] = oldWeight[tid];
			}
		}
		else
		{
			newDst[tid] = oldDst[tid];
			newWeight[tid] = oldWeight[tid];	
		}
		tid += ( blockDim.x * gridDim.x );
	}
	return;
}
