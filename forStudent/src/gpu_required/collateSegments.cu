/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void collateSegments_gpu(int * src, int * scanResult, int * output, int numEdges) {
	/*YOUR CODE HERE*/
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//this is the thread id and will be used as index
	while( tid < numEdges )
	{
		if( tid == (numEdges - 1) )//last element in the array
		{
			output[src[tid]] = scanResult[tid];
		}
		else if( src[tid] != src[tid+1] )
		{
			output[src[tid]] = scanResult[tid];
		}
		tid += ( blockDim.x * gridDim.x );
	}
	return;
}
