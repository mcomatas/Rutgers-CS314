/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
	/*YOUR CODE HERE*/
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//this is the thread id and will be used as index
	while( tid < numElements )
	{
		if( distance == 0 )//since it's exclusive prefix sum, when distance is 0 shift all values over to the right once
		{
			if( tid == 0 )
			{
				newSum[tid] = 0;
			}
			else
			{
				newSum[tid] = oldSum[tid - 1];
			}
		}
		else
		{
			if( tid - distance >= 0 )//if index - stride is still in the array, add those old sums together
			{
				newSum[tid] = oldSum[tid] + oldSum[tid - distance];
			}
			else//otherwise just move the element to the same index
			{
				newSum[tid] = oldSum[tid];
			}
		}
		tid += ( blockDim.x * gridDim.x );
	}
	return;
}
