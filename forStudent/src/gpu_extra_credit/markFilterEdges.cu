/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
	/*YOUR CODE HERE*/
	//basically check if source or dst is matched, then set to 1
	//if not matched then I can set to keepEdges[tid] = 0
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//this is the thread id and will be used as index
	while( tid < numEdges )
	{
		if( matches[src[tid]] == -1 && matches[dst[tid]] == -1 )//either source or destination has an unmatched node, meaning we want to keep this edge
		{
			keepEdges[tid] = 1;//if we want to keep the edge, set it to 1
		}
		else
		{
			keepEdges[tid] = 0;//otherwise set it to 0
		}
		tid += ( blockDim.x * gridDim.x );
	}
	return;
}
