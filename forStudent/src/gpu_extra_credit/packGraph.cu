/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void packGraph_gpu(int * newSrc, int * oldSrc, int * newDst, int * oldDst, int * newWeight, int * oldWeight, int * edgeMap, int numEdges) {
	/*YOUR CODE HERE*/
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//this is the thread id and will be used as index
	while( tid < numEdges )
	{
		if( edgeMap[tid + 1] != edgeMap[tid] )
		{
			newSrc[edgeMap[tid]] = oldSrc[tid];
			newDst[edgeMap[tid]] = oldDst[tid];
			newWeight[edgeMap[tid]] = oldWeight[tid];
		}
		tid += ( blockDim.x * gridDim.x );
	}
	return;
}
