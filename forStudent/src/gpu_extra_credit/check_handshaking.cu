/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	/*YOUR CODE HERE*/
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//this is the thread id and will be used as index
	while( tid < numNodes )//as long as thread id is within numNodes(array bounds)
	{
		if( tid == strongNeighbor[strongNeighbor[tid]] )
		{
			if( matches[tid] == -1 && matches[strongNeighbor[tid]] == -1 )//both need to equal -1, both nodes are unmatched
			{
				matches[tid] = strongNeighbor[tid];
				matches[strongNeighbor[tid]] = strongNeighbor[strongNeighbor[tid]];
			}
		}
		tid += ( blockDim.x * gridDim.x );
	}
	return;
}
